#include "hip/hip_runtime.h"
/*
 * GPUCauchyMoveGenerator.cpp
 *
 *  Created on: 30 sie 2019
 *      Author: pkua
 */

#include <hip/hip_math_constants.h>

#include "GPUCauchyMoveGenerator.h"

CUDA_DEV GPUCauchyMoveGenerator::GPUCauchyMoveGenerator(float sigma, unsigned int seed, size_t numberOfTrajectories) :
        sigma{sigma}, numberOfTrajectories{numberOfTrajectories} {
    this->states = new hiprandState[this->numberOfTrajectories];
    for (size_t i = 0; i < numberOfTrajectories; i++)
        hiprand_init(seed, i, 0, &(this->states[i]));
}

CUDA_DEV GPUCauchyMoveGenerator::~GPUCauchyMoveGenerator() {
    delete [] this->states;
}

CUDA_DEV float GPUCauchyMoveGenerator::randomCauchy() {
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    float uniform = hiprand_uniform(&(this->states[i]));
    return this->sigma * tanf(HIP_PI_F * (uniform - 0.5f));
}

CUDA_DEV Move GPUCauchyMoveGenerator::generateMove() {
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    float radius = this->randomCauchy();
    float angle = 2 * HIP_PI_F * hiprand_uniform(&(this->states[i]));

    return {radius * cosf(angle), radius * sinf(angle)};
}
