#include "hip/hip_runtime.h"
/*
 * GPUSimulationFactory.cpp
 *
 *  Created on: 26 sie 2019
 *      Author: pkua
 */

#include <sstream>
#include <fstream>
#include <vector>

#include "GPUSimulationFactory.h"
#include "utils/CudaCheck.h"
#include "move_generator/GPUGaussianMoveGenerator.h"
#include "move_filter/DefaultMoveFilter.h"
#include "move_filter/image_move_filter/ImageMoveFilter.h"
#include "move_filter/image_move_filter/WallBoundaryConditions.h"
#include "move_filter/image_move_filter/PeriodicBoundaryConditions.h"
#include "image/PPMImageReader.h"
#include "utils/Assertions.h"

namespace {
    enum MoveGeneratorType {
        GAUSSIAN,
        CAUCHY
    };

    enum MoveFilterType {
        DEFAULT,
        IMAGE
    };

    enum BoundaryConditionsType {
        WALL,
        PERIODIC
    };

    __global__
    void create_move_filter(unsigned long seed, size_t numberOfTrajectories, MoveFilterType moveFilterType,
                            uint32_t *intImageData, size_t width, size_t height,
                            BoundaryConditionsType boundaryConditionsType, MoveFilter **moveFilter,
                            ImageBoundaryConditions **boundaryConditions)
    {
        int i = blockIdx.x*blockDim.x + threadIdx.x;
        if (i != 0)
            return;

        if (moveFilterType == IMAGE) {
            if (boundaryConditionsType == WALL)
                (*boundaryConditions) = new WallBoundaryConditions();
            else if (boundaryConditionsType == PERIODIC)
                (*boundaryConditions) = new PeriodicBoundaryConditions();
            else
                (*boundaryConditions) = nullptr;
        } else {
            (*boundaryConditions) = nullptr;
        }

        if (moveFilterType == DEFAULT)
            (*moveFilter) = new DefaultMoveFilter();
        else if (moveFilterType == IMAGE)
            (*moveFilter) = new ImageMoveFilter(intImageData, width, height, *boundaryConditions, seed,
                                                numberOfTrajectories);
        else
            (*moveFilter) = nullptr;
    }



    __global__
    void create_move_generator(unsigned long seed, float sigma, size_t numberOfTrajectories,
                               MoveGeneratorType moveGeneratorType, MoveGenerator **moveGenerator)
    {
        int i = blockIdx.x*blockDim.x + threadIdx.x;
        if (i != 0)
            return;

        if (moveGeneratorType == GAUSSIAN)
            (*moveGenerator) = new GPUGaussianMoveGenerator(sigma, seed, numberOfTrajectories);
        else if (moveGeneratorType == CAUCHY)
            (*moveGenerator) = nullptr;
        else
            (*moveGenerator) = nullptr;
    }

    class MoveGeneratorOnGPU {
    private:
        MoveGeneratorType moveGeneratorType{};
        float sigma{};

    public:
        MoveGenerator *moveGenerator{};

        MoveGeneratorOnGPU(const Parameters &parameters) {
            std::istringstream moveGeneratorStream(parameters.moveGenerator);
            std::string moveGeneratorName;
            moveGeneratorStream >> moveGeneratorName >> this->sigma;
            if (!moveGeneratorStream)
                throw std::runtime_error("Malformed MoveGenerator parameters");
            Validate(this->sigma >= 0.f);

            if (moveGeneratorName == "GaussianMoveGenerator")
                this->moveGeneratorType = GAUSSIAN;
            else if (moveGeneratorName == "CauchyMoveGenerator")
                this->moveGeneratorType =  CAUCHY;
            else
                throw std::runtime_error("Unknown MoveGenerator: " + moveGeneratorName);
        }

        void allocateOnGPU(unsigned long seed, std::size_t numberOfWalks) {
            MoveGenerator **moveGeneratorPlaceholder{};
            cudaCheck( hipMalloc(&moveGeneratorPlaceholder, sizeof(MoveGenerator**)) );
            create_move_generator<<<1, 32>>>(seed, this->sigma, numberOfWalks, this->moveGeneratorType,
                                             moveGeneratorPlaceholder);
            cudaCheck( hipDeviceSynchronize() );
            cudaCheck( hipMemcpy(&(this->moveGenerator), moveGeneratorPlaceholder, sizeof(MoveGenerator*),
                                  hipMemcpyDeviceToHost) );
            cudaCheck( hipFree(moveGeneratorPlaceholder) );
        }
    };

    class MoveFilterOnGPU {
    private:
        MoveFilterType moveFilterType{};
        BoundaryConditionsType boundaryConditionsType{};
        Image image{};

        void fetchImageData(std::istringstream &moveFilterStream, std::ostream &logger) {
            std::string imageFilename;
            moveFilterStream >> imageFilename;
            if (!moveFilterStream)
                throw std::runtime_error("Malformed ImageMoveFilter parameters");

            std::ifstream imageFile(imageFilename);
            if (!imageFile)
                throw std::runtime_error("Cannot open " + imageFilename + " to load image");

            PPMImageReader imageReader;
            this->image = imageReader.read(imageFile);
            logger << "[GPUSimulationFactory] Loaded image " << imageFilename << " (" << this->image.getWidth();
            logger << "px x " << this->image.getHeight() << "px)" << std::endl;
        }

        void fetchBoundaryConditions(std::istringstream &moveFilterStream) {
            std::string imageBCType;
            moveFilterStream >> imageBCType;
            if (!moveFilterStream)
                throw std::runtime_error("Malformed ImageMoveFilter parameters");

            if (imageBCType == "WallBoundaryConditions")
                this->boundaryConditionsType = WALL;
            else if (imageBCType == "PeriodicBoundaryConditions")
                this->boundaryConditionsType = PERIODIC;
            else
                throw std::runtime_error("Unknown ImageBoundaryConditions: " + imageBCType);
        }

    public:
        MoveFilter *moveFilter{};
        ImageBoundaryConditions *boundaryConditions{};

        MoveFilterOnGPU(const Parameters &parameters, std::ostream &logger) {
            std::istringstream moveFilterStream(parameters.moveFilter);
            std::string moveFilterName;
            moveFilterStream >> moveFilterName;

            if (moveFilterName == "DefaultMoveFilter")
                this->moveFilterType = DEFAULT;
            else if (moveFilterName == "ImageMoveFilter")
                this->moveFilterType = IMAGE;
            else
                throw std::runtime_error("Unknown MoveFilter: " + moveFilterName);

            if (this->moveFilterType == IMAGE) {
                this->fetchImageData(moveFilterStream, logger);
                this->fetchBoundaryConditions(moveFilterStream);
            }
        }

        void allocateOnGPU(unsigned long seed, std::size_t numberOfWalks) {
            MoveFilter **moveFilterPlaceholder{};
            ImageBoundaryConditions **boundaryConditionsPlaceholder{};
            uint32_t *gpuIntImageData{};

            cudaCheck( hipMalloc(&moveFilterPlaceholder, sizeof(MoveFilter**)) );
            cudaCheck( hipMalloc(&boundaryConditionsPlaceholder, sizeof(ImageBoundaryConditions**)) );

            auto intImageData = this->image.getIntData();
            if (this->moveFilterType == IMAGE) {
                cudaCheck( hipMalloc(&gpuIntImageData, intImageData.size()*sizeof(uint32_t)));
                cudaCheck( hipMemcpy(gpuIntImageData, intImageData.data(), intImageData.size()*sizeof(uint32_t),
                                      hipMemcpyHostToDevice) );
            }

            create_move_filter<<<1, 32>>>(seed, numberOfWalks, this->moveFilterType, gpuIntImageData,
                                          this->image.getWidth(), this->image.getHeight(), this->boundaryConditionsType,
                                          moveFilterPlaceholder, boundaryConditionsPlaceholder);
            cudaCheck( hipDeviceSynchronize() );

            cudaCheck( hipMemcpy(&(this->moveFilter), moveFilterPlaceholder, sizeof(MoveFilter*),
                                  hipMemcpyDeviceToHost) );
            cudaCheck( hipMemcpy(&(this->boundaryConditions), boundaryConditionsPlaceholder,
                                  sizeof(ImageBoundaryConditions*), hipMemcpyDeviceToHost) );

            cudaCheck( hipFree(moveFilterPlaceholder) );
            cudaCheck( hipFree(boundaryConditionsPlaceholder) );
            cudaCheck( hipFree(gpuIntImageData) );
        }
    };
}


__global__
void delete_objects(MoveGenerator *moveGenerator, MoveFilter *moveFilter, ImageBoundaryConditions *boundaryConditions)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i != 0)
        return;

    delete moveGenerator;
    delete moveFilter;
    delete boundaryConditions;
}

void GPUSimulationFactory::initializeSeedGenerator(const Parameters &parameters, std::ostream &logger) {
    if (parameters.seed == "random") {
        unsigned long randomSeed = std::random_device()();
        this->seedGenerator.seed(randomSeed);
        logger << "[GPUSimulationFactory] Using random seed: " << randomSeed << std::endl;
    } else {
        this->seedGenerator.seed(std::stoul(parameters.seed));
    }
}

GPUSimulationFactory::GPUSimulationFactory(const Parameters& parameters, std::ostream& logger) {
    this->initializeSeedGenerator(parameters, logger);

    MoveGeneratorOnGPU gpuMoveGenerator(parameters);
    MoveFilterOnGPU gpuMoveFilter(parameters, logger);

    gpuMoveGenerator.allocateOnGPU(this->seedGenerator(), parameters.numberOfWalks);
    gpuMoveFilter.allocateOnGPU(this->seedGenerator(), parameters.numberOfWalks);

    this->moveGenerator = gpuMoveGenerator.moveGenerator;
    this->moveFilter = gpuMoveFilter.moveFilter;
    this->imageBoundaryConditions = gpuMoveFilter.boundaryConditions;

    Move drift = {parameters.driftX, parameters.driftY};
    this->randomWalker.reset(new GPURandomWalker(parameters.numberOfWalks, parameters.numberOfSteps,
                                                 parameters.tracerRadius, drift, this->moveGenerator,
                                                 this->moveFilter));
}

GPUSimulationFactory::~GPUSimulationFactory() {
    delete_objects<<<1, 32>>>(this->moveGenerator, this->moveFilter, this->imageBoundaryConditions);
    cudaCheck( hipDeviceSynchronize() );
}

RandomWalker& GPUSimulationFactory::getRandomWalker() {
    return *this->randomWalker;
}
