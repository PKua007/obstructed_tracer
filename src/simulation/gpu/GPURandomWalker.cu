#include "hip/hip_runtime.h"
/*
 * GPURandomWalker.cpp
 *
 *  Created on: 26 sie 2019
 *      Author: pkua
 */

#include <stdexcept>
#include <ostream>

#include "GPURandomWalker.h"
#include "utils/Assertions.h"
#include "utils/CudaCheck.h"
#include "simulation/SimulationTimer.h"


__global__
void gpu_random_walk(size_t numberOfTrajectories, RandomWalker::WalkParameters walkParameters,
                     MoveGenerator* moveGenerator, MoveFilter* moveFilter, Point **trajectories,
                     size_t *acceptedSteps)
{
    int i = CUDA_THREAD_IDX;
    if (i >= numberOfTrajectories)
        return;

    Tracer tracer = moveFilter->randomValidTracer();
    trajectories[i][0] = tracer.getPosition();

    acceptedSteps[i] = 0;
    for (size_t step = 1; step <= walkParameters.numberOfSteps; step++) {
        Move move = moveGenerator->generateMove() + walkParameters.drift;
        if (moveFilter->isMoveValid(tracer, move)) {
            tracer += move;
            trajectories[i][step] = tracer.getPosition();
            acceptedSteps[i]++;
        } else {
            trajectories[i][step] = tracer.getPosition();
        }
    }
}

__global__
void setup_move_filter(MoveFilter* moveFilter, float tracerRadius) {
    moveFilter->setupForTracerRadius(tracerRadius);
}

GPURandomWalker::TrajectoriesOnGPU::TrajectoriesOnGPU(std::size_t numberOfTrajectories, std::size_t numberOfSteps) :
        numberOfTrajectories{numberOfTrajectories}, numberOfSteps{numberOfSteps},
        cpuVectorOfGPUTrajectories(numberOfTrajectories), cpuVectorOfAcceptedSteps(numberOfTrajectories)
{
    cudaCheck( hipMalloc(&this->gpuArrayOfGPUTrajectories, this->numberOfTrajectories*sizeof(Point*)) );
    cudaCheck( hipMalloc(&this->gpuArrayOfAcceptedSteps, this->numberOfTrajectories*sizeof(size_t)) );

    for (std::size_t i = 0; i < this->numberOfTrajectories; i++) {
        // Number of steps plus ONE STEP for initial tracer
        cudaCheck( hipMalloc(&(this->cpuVectorOfGPUTrajectories[i]),
                              (this->numberOfSteps + 1) * sizeof(Point)) );
    }
    cudaCheck( hipMemcpy(this->gpuArrayOfGPUTrajectories, this->cpuVectorOfGPUTrajectories.data(),
                          this->numberOfTrajectories*sizeof(Point*), hipMemcpyHostToDevice) );
}

GPURandomWalker::TrajectoriesOnGPU::~TrajectoriesOnGPU() {
    cudaCheck( hipFree(this->gpuArrayOfGPUTrajectories) );
    for (auto gpuTrajectory : cpuVectorOfGPUTrajectories)
        cudaCheck( hipFree(gpuTrajectory) );
    cudaCheck( hipFree(this->gpuArrayOfAcceptedSteps) );
}


void GPURandomWalker::TrajectoriesOnGPU::copyToCPU(std::vector<GPUTrajectory> &trajectories) {
    cudaCheck( hipMemcpy(this->cpuVectorOfAcceptedSteps.data(), this->gpuArrayOfAcceptedSteps,
                          this->numberOfTrajectories*sizeof(size_t), hipMemcpyDeviceToHost) );

    for (std::size_t i = 0; i < this->numberOfTrajectories; i++) {
        trajectories[i].copyGPUData(this->cpuVectorOfGPUTrajectories[i], this->numberOfSteps + 1,
                                    this->cpuVectorOfAcceptedSteps[i]);
    }
}

GPURandomWalker::GPURandomWalker(std::size_t numberOfTrajectories, RandomWalker::WalkParameters walkParameters,
                                 std::size_t numberOfMoveFilterSetupThreads,  MoveGenerator* moveGenerator,
                                 MoveFilter* moveFilter, std::ostream &logger) :
        numberOfTrajectories{numberOfTrajectories}, walkParameters{walkParameters},
        numberOfMoveFilterSetupThreads{numberOfMoveFilterSetupThreads}, moveGenerator{moveGenerator},
        moveFilter{moveFilter}, trajectoriesOnGPU(numberOfTrajectories, walkParameters.numberOfSteps)
{
    Expects(numberOfTrajectories > 0);
    Expects(numberOfSteps > 0);
    Expects(tracerRadius >= 0.f);

    this->trajectories.resize(numberOfTrajectories);
    this->setupMoveFilterForTracerRadius(logger);
}

void GPURandomWalker::setupMoveFilterForTracerRadius(std::ostream& logger) {
    int numberOfBlocks = (this->numberOfMoveFilterSetupThreads + blockSize - 1)
            / blockSize;
    logger << "[GPURandomWalker::run] Setting up MoveFilter... " << std::flush;
    setup_move_filter<<<numberOfBlocks, blockSize>>>(this->moveFilter, this->walkParameters.tracerRadius);
    cudaCheck(hipDeviceSynchronize());
    logger << "completed." << std::endl;
}

void GPURandomWalker::run(std::ostream& logger) {
    SimulationTimer timer(this->numberOfTrajectories);
    timer.start();

    logger << "[GPURandomWalker::run] Starting simulation... " << std::flush;
    int numberOfBlocks = (numberOfTrajectories + blockSize - 1) / blockSize;
    gpu_random_walk<<<numberOfBlocks, blockSize>>>(this->numberOfTrajectories, this->walkParameters,
                                                   this->moveGenerator, this->moveFilter,
                                                   trajectoriesOnGPU.getTrajectoriesArray(),
                                                   trajectoriesOnGPU.getAcceptedStepsArray());
    cudaCheck( hipDeviceSynchronize() );
    logger << "completed." << std::endl;

    logger << "[GPURandomWalker::run] Fetching data from video memory... " << std::flush;
    trajectoriesOnGPU.copyToCPU(this->trajectories);
    logger << "completed." << std::endl;

    timer.stop();
    timer.showInfo(logger);
}

std::size_t GPURandomWalker::getNumberOfTrajectories() const {
    return this->numberOfTrajectories;
}

const Trajectory &GPURandomWalker::getTrajectory(std::size_t index) const {
    return this->trajectories[index];
}
