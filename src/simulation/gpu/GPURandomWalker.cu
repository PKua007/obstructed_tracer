#include "hip/hip_runtime.h"
/*
 * GPURandomWalker.cpp
 *
 *  Created on: 26 sie 2019
 *      Author: pkua
 */

#include <stdexcept>
#include <ostream>

#include "GPURandomWalker.h"
#include "utils/Assertions.h"
#include "utils/CudaCheck.h"

__global__
void gpu_random_walk(size_t numberOfTrajectories, size_t numberOfSteps, float tracerRadius, Move drift,
                     MoveGenerator* moveGenerator, MoveFilter* moveFilter, Point **trajectories, size_t *acceptedSteps)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= numberOfTrajectories)
        return;

    Tracer tracer = moveFilter->randomValidTracer(tracerRadius);
    trajectories[i][0] = tracer.getPosition();

    acceptedSteps[i] = 0;
    for (size_t step = 1; step <= numberOfSteps; step++) {
        Move move = moveGenerator->generateMove() + drift;
        if (moveFilter->isMoveValid(tracer, move)) {
            tracer += move;
            trajectories[i][step] = tracer.getPosition();
            acceptedSteps[i]++;
        } else {
            trajectories[i][step] = tracer.getPosition();
        }
    }
}

GPURandomWalker::GPURandomWalker(std::size_t numberOfTrajectories, std::size_t numberOfSteps, float tracerRadius,
                                 Move drift, MoveGenerator* moveGenerator, MoveFilter* moveFilter) :
        numberOfSteps{numberOfSteps}, tracerRadius{tracerRadius}, drift{drift}, moveGenerator{moveGenerator},
        moveFilter{moveFilter} {
    Expects(numberOfTrajectories > 0);
    Expects(numberOfSteps > 0);
    Expects(tracerRadius >= 0.f);
    this->trajectories.resize(numberOfTrajectories);
}

void GPURandomWalker::run(std::ostream& logger) {
    std::size_t numberOfTrajectories = this->trajectories.size();

    Point **gpuTrajectories;
    size_t *gpuAcceptedSteps;
    cudaCheck( hipMalloc(&gpuTrajectories, numberOfTrajectories*sizeof(Point*)) );
    cudaCheck( hipMalloc(&gpuAcceptedSteps, numberOfTrajectories*sizeof(size_t)) );

    std::vector<Point*> cpuTrajectoryPointers(numberOfTrajectories);
    for (std::size_t i = 0; i < numberOfTrajectories; i++)
        cudaCheck( hipMalloc(&(cpuTrajectoryPointers[i]), (this->numberOfSteps + 1) * sizeof(Point)) );
    cudaCheck( hipMemcpy(gpuTrajectories, cpuTrajectoryPointers.data(), numberOfTrajectories*sizeof(Point*),
                          hipMemcpyHostToDevice) );

    logger << "[GPURandomWalker::run] Starting simulation... " << std::flush;

    int blockSize = 32;
    int numberOfBlocks = (numberOfTrajectories + blockSize - 1) / blockSize;
    gpu_random_walk<<<numberOfBlocks, blockSize>>>(numberOfTrajectories, this->numberOfSteps, this->tracerRadius,
                                                   this->drift, this->moveGenerator, this->moveFilter, gpuTrajectories,
                                                   gpuAcceptedSteps);

    cudaCheck( hipPeekAtLastError() );

    logger << "completed." << std::endl;

    cudaCheck( hipFree(gpuTrajectories) );

    std::vector<size_t> cpuAcceptedSteps(numberOfTrajectories);
    cudaCheck( hipMemcpy(cpuAcceptedSteps.data(), gpuAcceptedSteps, numberOfTrajectories*sizeof(size_t),
                          hipMemcpyDeviceToHost) );
    cudaCheck( hipFree(gpuAcceptedSteps) );

    for (std::size_t i = 0; i < numberOfTrajectories; i++)
        this->trajectories[i].moveGPUData(cpuTrajectoryPointers[i], (this->numberOfSteps + 1), cpuAcceptedSteps[i]);
}

std::size_t GPURandomWalker::getNumberOfTrajectories() const {
    return this->trajectories.size();
}

const Trajectory &GPURandomWalker::getTrajectory(std::size_t index) const {
    return this->trajectories[index];
}
