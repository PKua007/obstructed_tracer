#include "hip/hip_runtime.h"
/*
 * GPUGaussianMoveGenerator.cpp
 *
 *  Created on: 26 sie 2019
 *      Author: pkua
 */

#include "GPUGaussianMoveGenerator.h"

CUDA_DEV GPUGaussianMoveGenerator::GPUGaussianMoveGenerator(float sigma, unsigned int seed,
                                                            size_t numberOfTrajectories)
        // Divide sigma by sqrt(2), because if we sample x and y with sigma^2, then r is sampled from 2sigma^2
        : sigma{sigma * float{M_SQRT1_2}}, numberOfTrajectories{numberOfTrajectories}
{
    this->states = new hiprandState[this->numberOfTrajectories];
    for (size_t i = 0; i < numberOfTrajectories; i++)
        hiprand_init(seed, i, 0, &(this->states[i]));
}

CUDA_DEV GPUGaussianMoveGenerator::~GPUGaussianMoveGenerator() {
    delete [] this->states;
}

CUDA_DEV Move GPUGaussianMoveGenerator::generateMove() {
    int i = CUDA_THREAD_IDX;

    return {hiprand_normal(&(this->states[i])) * this->sigma, hiprand_normal(&(this->states[i])) * this->sigma};
}
