#include "hip/hip_runtime.h"
/*
 * ImageMoveFilter.cpp
 *
 *  Created on: 29 lip 2019
 *      Author: pkua
 */

#include <iostream>

#include "ImageMoveFilter.h"
#include "utils/Assertions.h"
#include "utils/Utils.h"

namespace {
    struct ImageMove {
        int x{};
        int y{};

        CUDA_HOSTDEV ImageMove(int x, int y) : x{x}, y{y} { };
    };

    CUDA_HOSTDEV ImageMove operator-(ImagePoint p1, ImagePoint p2) {
        return {p1.x - p2.x, p1.y - p2.y};
    }

    template <typename T> CUDA_HOSTDEV int sgn(T val) {
        return (T(0) < val) - (val < T(0));
    }
}


ImageMoveFilter::ImageMoveFilter(unsigned int *intImageData, size_t width, size_t height,
                                 ImageBoundaryConditions *imageBC, unsigned long seed, size_t numberOfTrajectories) :
        width{width}, height{height}, imageBC{imageBC} {
    this->initializeGenerators(seed, numberOfTrajectories);

    this->validPointsMapSize = this->width * this->height;
    this->validPointsMap = new bool[this->validPointsMapSize];
    this->validTracersMap = new bool[this->validPointsMapSize];

    // On CPU in case of allocation fail std::bad_alloc should be thrown, we only need to check on GPU
    #if CUDA_DEVICE_COMPILATION
        if (this->validPointsMap == nullptr || this->validTracersMap == nullptr) {
            printf("[ImageMoveFilter] Allocation of arrays of valid points failed. Increase the size of GPU heap.\n");
            asm("trap;");
        }
    #endif

    this->imageBC->setupDimensions(this->width, this->height);

    // Image y axis starts from left upper corner downwards, so image is scanned from the bottom left, because
    // validPointsMap is in "normal" coordinate system, with (0, 0) in left bottom corner
    size_t i = 0;
    for (size_t y = 0; y < this->height; y++) {
        for (size_t x = 0; x < this->width; x++) {
            ImagePoint imagePoint = {static_cast<int>(x), static_cast<int>(this->height - y - 1)};
            if (intImageData[this->pointToIndex(imagePoint)] == 0xffffffff)
                this->validPointsMap[i] = true;
            else
                this->validPointsMap[i] = false;

            i++;
        }
    }

    // CPU version should be in valid state - for tracer radius 0 - even without calling setupForTracerRadius.
    // GPU version spares some time not doing it
    #if CUDA_HOST_COMPILATION
        this->setupForTracerRadius(0.f);
    #endif
}

ImageMoveFilter::~ImageMoveFilter() {
    delete [] this->validPointsMap;
    delete [] this->validTracersMap;
    #if CUDA_DEVICE_COMPILATION
        delete [] this->states;
    #endif
}


#if CUDA_DEVICE_COMPILATION

    void ImageMoveFilter::initializeGenerators(unsigned long seed, size_t numberOfTrajectories) {
        this->states = new hiprandState[numberOfTrajectories];
        for (size_t i = 0; i < numberOfTrajectories; i++)
            hiprand_init(seed, i, 0, &(this->states[i]));
    }

#else // CUDA_HOST_COMPILATION

    void ImageMoveFilter::initializeGenerators(unsigned long seed, size_t numberOfTrajectories) {
        this->randomGenerator.seed(seed);
    }

#endif


bool ImageMoveFilter::checkValidPointsMap(ImagePoint point) const {
    point = this->imageBC->applyOnImagePoint(point);
    return this->validPointsMap[this->pointToIndex(point)];
}

bool ImageMoveFilter::checkValidTracersMap(ImagePoint point) const {
    if (!this->imageBC->isImagePointInBounds(point, this->tracerRadius))
        return false;

    point = this->imageBC->applyOnImagePoint(point);
    return this->validTracersMap[this->pointToIndex(point)];
}

bool ImageMoveFilter::isPointValid(ImagePoint point, float pointRadius) const {
    Expects(pointRadius >= 0.f);

    int intPointRadius = static_cast<int>(pointRadius);
    if (!this->imageBC->isImagePointInBounds(point, intPointRadius))
        return false;

    if (pointRadius == 0.f)
        return this->checkValidPointsMap(point);

    for (int x = -intPointRadius; x <= intPointRadius; x++) {
        for (int y = -intPointRadius; y <= intPointRadius; y++) {
            if (x*x + y*y > pointRadius*pointRadius)
                continue;

            if (!this->checkValidPointsMap({point.x + x, point.y + y}))
                return false;
        }
    }
    return true;
}

bool ImageMoveFilter::isLineValid(ImagePoint from, ImagePoint to) const {
    ImageMove imageMove = to - from;
    if (abs(imageMove.x) > abs(imageMove.y)) {
        float a = float(imageMove.y) / float(imageMove.x);
        for (int x = from.x; x != to.x; x += sgn(imageMove.x)) {
            int y = static_cast<int>(round(from.y + a * (x - from.x)));
            if (!this->checkValidTracersMap({ x, y }))
                return false;
        }
    } else {
        float a = float(imageMove.x) / float(imageMove.y);
        for (int y = from.y; y != to.y; y += sgn(imageMove.y)) {
            int x = static_cast<int>(round(from.x + a * (y - from.y)));
            if (!this->checkValidTracersMap({ x, y }))
                return false;
        }
    }
    return true;
}

ImagePoint ImageMoveFilter::indexToImagePoint(size_t index) const {
    Expects(index < this->validPointsMapSize);
    return {static_cast<int>(index % this->width), static_cast<int>(index / this->width)};
}

size_t ImageMoveFilter::pointToIndex(ImagePoint point) const {
    return point.x + this->width * point.y;
}

#if CUDA_DEVICE_COMPILATION

    float ImageMoveFilter::randomUniformNumber() {
        // 1 minus hiprand_normal, because it samples from (0, 1], and we want [0, 1)
        return 1.f - hiprand_uniform(&(this->states[CUDA_THREAD_IDX]));
    }

#else // CUDA_HOST_COMPILATION

    float ImageMoveFilter::randomUniformNumber() {
        return this->uniformDistribution(this->randomGenerator);
    }

#endif

#if CUDA_DEVICE_COMPILATION

    ImagePoint ImageMoveFilter::randomTracerImagePosition() {
        ImagePoint imagePosition;
        do {
            float floatMapIndex = this->randomUniformNumber() * this->validPointsMapSize;
            size_t mapIndex = static_cast<size_t>(floatMapIndex);
            imagePosition = this->indexToImagePoint(mapIndex);
        } while(!this->checkValidTracersMap(imagePosition));
        return imagePosition;
    }

#else // CUDA_HOST_COMPILATION

    ImagePoint ImageMoveFilter::randomTracerImagePosition() {
        float floatCacheIndex = this->randomUniformNumber() * this->validTracerIndicesCache.size();
        size_t cacheIndex = static_cast<size_t>(floatCacheIndex);
        Assert(cacheIndex < this->validTracerIndicesCache.size());
        size_t tracerIndex = this->validTracerIndicesCache[cacheIndex];
        return this->indexToImagePoint(tracerIndex);
    }

#endif

bool ImageMoveFilter::isMoveValid(Tracer tracer, Move move) const {
    Point from = tracer.getPosition();
    Point to = from + move;
    ImagePoint imageFrom(from);
    ImagePoint imageTo(to);

    if (imageFrom == imageTo)
        return true;

    if (!checkValidTracersMap(imageTo))
        return false;

    return isLineValid(imageFrom, imageTo);
}

Tracer ImageMoveFilter::randomValidTracer() {
    ImagePoint imagePosition = this->randomTracerImagePosition();
    float pixelOffsetX = this->randomUniformNumber();
    float pixelOffsetY = this->randomUniformNumber();

    Point tracerPosition = {imagePosition.x + pixelOffsetX, imagePosition.y + pixelOffsetY};
    return Tracer(tracerPosition, this->tracerRadius);
}

#if CUDA_DEVICE_COMPILATION

    void ImageMoveFilter::setupForTracerRadius(float radius) {
        int i = CUDA_THREAD_IDX;
        if (i >= this->validPointsMapSize)
            return;

        this->tracerRadius = radius;
        this->validTracersMap[i] = this->isPointValid(this->indexToImagePoint(i), radius);
    }

#else // CUDA_HOST_COMPILATION

    void ImageMoveFilter::setupForTracerRadius(float radius) {
        Expects(radius >= 0.f);
        this->tracerRadius = radius;

        this->validTracerIndicesCache.clear();
        for (size_t i = 0; i < this->validPointsMapSize; i++) {
            if (this->isPointValid(this->indexToImagePoint(i), radius)) {
                this->validTracersMap[i] = true;
                this->validTracerIndicesCache.push_back(i);
            } else {
                this->validTracersMap[i] = false;
            }
        }

        if (this->validTracerIndicesCache.empty())
            throw std::runtime_error("No valid points found in a given image");
    }

#endif


size_t ImageMoveFilter::getNumberOfAllPoints() const {
    return this->validPointsMapSize;
}

#if CUDA_HOST_COMPILATION

    size_t ImageMoveFilter::getNumberOfValidTracers() {
        return this->validTracerIndicesCache.size();
    }

#endif /* HOST_COMPILATION */
